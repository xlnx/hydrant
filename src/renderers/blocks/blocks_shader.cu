#include "hip/hip_runtime.h"
#include "blocks_shader.hpp"

struct VisBlocksShaderKernel : BlocksShader
{
	__host__ __device__ int
	  skip_nblock_steps( Ray &ray, vec3 const &ip,
						 int nblocks, float cdu, float step ) const
	{
		float tnear, tfar;
		ray.intersect( Box3D{ ip, ip + 1.f }, tnear, tfar );
		float di = ceil( ( tfar + ( nblocks - 1 ) * cdu ) / step );
		ray.o += ray.d * di * step;
		return (int)di;
	}

	__host__ __device__ void
	  main( Pixel &pixel_in_out ) const
	{
		const auto cdu = 1.f / compMax( abs( pixel_in_out.ray.d ) );
		const auto opacity_threshold = 0.95f;

		auto pixel = pixel_in_out;
		auto &ray = pixel.ray;
		auto &nsteps = pixel.nsteps;

		while ( nsteps > 0 ) {
			vec3 ip = floor( ray.o );
			if ( int cd = chebyshev.sample_3d<int>( ip ) ) {
				nsteps -= skip_nblock_steps( ray, ip, cd, cdu, step );
			} else {
				auto rip = clamp( ip, bbox.min, bbox.max - 1.f ) / ( bbox.max - bbox.min );
				if ( render_mode == BlocksRenderMode::Volume ) {
					auto mean = mean_tex.sample_3d<float>( ip );
					auto col = vec4( rip, 1.f ) * density * mean;
					pixel.v += col * ( 1.f - pixel.v.w );
					if ( pixel.v.w > opacity_threshold ) {
						nsteps = 0;
						break;
					}
				} else if ( render_mode == BlocksRenderMode::Solid ) {
					pixel.v = vec4( rip, 1.f );
					nsteps = 0;
					break;
				}
			}
			ray.o += ray.d * step;
			nsteps -= 1;
		}
		pixel_in_out = pixel;
	}
};

REGISTER_SHADER_BUILDER(
  name( "vis_blocks_shader" )
	.cuda<VisBlocksShaderKernel>()
	.cpu<VisBlocksShaderKernel>(),
  BlocksShader );
